#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <algorithm>
#include <filesystem>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

using namespace std;
namespace fs = filesystem;

struct movie {
    int index;
    int start;
    int end;
    int category;
};

struct rbitset {
    int size = 24;
    int bs[24];

    __host__ __device__
    rbitset() {
        for (int i = 0; i < size; i++) {
            bs[i] = 0;
        }
    }

    __host__ __device__
    int get(int index) const {
        return bs[index];
    }

    __host__ __device__
    void set(int index) {
        bs[index] = 1;
    }
};

void writeOutput(string filename, int nMovies) {
    fs::path dir_path("outputs/");
    if (!fs::exists(dir_path)) {
        fs::create_directory(dir_path);
    }

    ofstream inputFile;
    filename = filename.replace(0, 5, "outputs/output-exaustiva-thrust");
    inputFile.open(filename);
    inputFile << nMovies << endl;

    inputFile.close();
}

struct orderByEndTime {
    __host__ __device__
    bool operator()(const movie& a, const movie& b) const {
        if (a.end != b.end) {
            return a.end < b.end;
        }
        return a.start < b.start;
    }
};

void printMovies(vector<movie> movies) {
    for (int i = 0; i < movies.size(); i++) {
        cout << movies[i].index << " " << movies[i].start << " " << movies[i].end << " " << movies[i].category + 1 << endl;
    }
}

struct populateBitset {
    __host__ __device__
    void operator()(int start, int end, rbitset& bitset) {
        if (start == 0) {
            for (int i = start; i < end; i++) {
                bitset.set(i);
            }
        } else {
            for (int i = start - 1; i < end; i++) {
                bitset.set(i);
            }
        }
    }
};

struct isWatchable {
    __host__ __device__
    bool operator()(int start, int end, const rbitset& bitset) const {
        if (start == end) {
            return false;
        }

        if (start > end) {
            return false;
        }

        for (int i = start; i < end; i++) {
            if (bitset.get(i)) {
                return false;
            }
        }

        return true;
    }
};

struct searchMoviesGpu {
    int numMovies;
    int numCat;
    movie* movies;
    int* categories;

    searchMoviesGpu(int _numMovies, int _numCat, movie* _movies, int* _categories) : 
        numMovies(_numMovies),
        numCat(_numCat),
        movies(_movies),
        categories(_categories) {}

    __host__ __device__
    int operator()(const int& combination) const {
        int nMovies = 0;
        rbitset bitset;
        int tempCategories[20];
        for (int i = 0; i < numCat; i++) {
            tempCategories[i] = *(categories+i);
        }

        // Max combinations without using pow() because it is super unefficient
        for (int i = 0; i < numMovies; i++) {
            if (combination & (1 << i)) {
                movie& movieRef = movies[i];
                if (tempCategories[movieRef.category] != 0 && isWatchable()(movieRef.start, movieRef.end, bitset)) {
                    populateBitset()(movieRef.start, movieRef.end, bitset);
                    nMovies++;
                    tempCategories[movieRef.category]--;
                }
            }
        }

        return nMovies;
    }
};

int main(int argc, char* argv[]) {
    int numMovies;
    int numCategories;

    string filename = argv[1];

    thrust::host_vector<int> categories;
    thrust::host_vector<movie> movies;

    cin >> numMovies >> numCategories;

    for (int i = 0; i < numCategories; i++) {
        int num;
        cin >> num;
        categories.push_back(num);
    }

    for (int i = 0; i < numMovies; i++) {
        int start, end, cat;
        cin >> start >> end >> cat;

        if (end == 0) {
            end = 24;
        }

        movie m = {i+1, start, end, cat-1};
        movies.push_back(m);
    }

    thrust::sort(thrust::host, movies.begin(), movies.end(), orderByEndTime());

    thrust::device_vector<int> combinations((1 << numMovies));
    thrust::sequence(combinations.begin(), combinations.end());

    thrust::device_vector<int> chosenMovies((1 << numMovies));
    thrust::device_vector<movie> movies_gpu(movies);
    thrust::device_vector<int> categories_gpu(categories);

    thrust::transform(
        combinations.begin(), 
        combinations.end(), 
        chosenMovies.begin(), 
        searchMoviesGpu(numMovies, numCategories, raw_pointer_cast(movies_gpu.data()), raw_pointer_cast(categories_gpu.data()))
    );

    thrust::host_vector<int> result = chosenMovies;

    int maxMovies = 0;
    for (long int i = 0; i < (1 << numMovies); i++) {
        if (result[i] > maxMovies) {
            maxMovies = result[i];
        }
    }

    writeOutput(filename, maxMovies);

    return maxMovies;
}
